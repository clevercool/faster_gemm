#include "hip/hip_runtime.h"
#include <assert.h>
#include <stdio.h>
#include <hiprand.h>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <mma.h>
using namespace nvcuda;

#include "tzgemm_mix.cu"

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
    if (stat != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
    }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}

#define checkKernelErrors(expr)                                   \
    do                                                            \
    {                                                             \
        expr;                                                     \
                                                                  \
        hipError_t __err = hipGetLastError();                   \
        if (__err != hipSuccess)                                 \
        {                                                         \
            printf("Line %d: '%s' failed: %s\n", __LINE__, #expr, \
                   hipGetErrorString(__err));                    \
            abort();                                              \
        }                                                         \
    } while (0)


__host__ void init_host_matrices(float *a, float *b, 
    int M_GLOBAL, int N_GLOBAL, int K_GLOBAL) {
    for (int i = 0; i < M_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            a[i * K_GLOBAL + j] = (float)(rand() % 3);
        }
    }

    for (int i = 0; i < N_GLOBAL; i++) {
        for (int j = 0; j < K_GLOBAL; j++) {
            b[i * K_GLOBAL + j] = (float)(rand() % 3);
        }
    }
}
        

int main(int argc, char* argv[]) {
    int wmma_iter = 1;
    int M_INPUT = 16 * 32 * 8;
	int N_INPUT = 16 * 4 * 48;
	int K_INPUT = 16 * 4 * 12;
    if (argc == 5) {
        wmma_iter = atoi(argv[1]);
        M_INPUT = atoi(argv[2]);
        N_INPUT = atoi(argv[3]);
        K_INPUT = atoi(argv[4]);
    } 

    int M_GLOBAL = (M_INPUT < 64) ? 64 : (M_INPUT / 64) * 64;
	int N_GLOBAL = (N_INPUT < 64) ? 64 : (N_INPUT / 64) * 64;
	int K_GLOBAL = (K_INPUT < 64) ? 64 : (K_INPUT / 64) * 64;

	int M_TILES = M_GLOBAL / WMMA_M;
	int N_TILES = N_GLOBAL / WMMA_N;
	int K_TILES = K_GLOBAL / WMMA_K;

    float kernel_time;
    hiprandGenerator_t gen;
    hipEvent_t startKERNEL;
    hipEvent_t stopKERNEL;
    cudaErrCheck(hipEventCreate(&startKERNEL));
    cudaErrCheck(hipEventCreate(&stopKERNEL));

    // wmma variables
    // ----------------------------------------------------------------------------------------------------------------------
    float *ori_host_A = NULL;
	float *ori_host_B = NULL;
	float *ori_result_C = NULL;
    float *mix_result_C = NULL;
    
	float *ori_device_A = NULL;
	float *ori_device_B = NULL;

	half *ori_wmma_A = NULL;
	half *ori_wmma_B = NULL;
	float *ori_wmma_C = NULL;

    half *mix_wmma_A = NULL;
	half *mix_wmma_B = NULL;
	float *mix_wmma_C = NULL;

	ori_host_A = (float *)malloc(sizeof(float) * M_GLOBAL * K_GLOBAL);
	ori_host_B = (float *)malloc(sizeof(float) * K_GLOBAL * N_GLOBAL);
	ori_result_C = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);
	mix_result_C = (float *)malloc(sizeof(float) * M_GLOBAL * N_GLOBAL);

    init_host_matrices(ori_host_A, ori_host_B, M_GLOBAL, N_GLOBAL, K_GLOBAL);
    
    curandErrCheck(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(gen, 1337ULL));

	cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_wmma_A), sizeof(half) * M_GLOBAL * K_GLOBAL));
	cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_wmma_B), sizeof(half) * N_GLOBAL * K_GLOBAL));
	cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_wmma_C), sizeof(float) * M_GLOBAL * N_GLOBAL));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&mix_wmma_A), sizeof(half) * M_GLOBAL * K_GLOBAL));
	cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&mix_wmma_B), sizeof(half) * N_GLOBAL * K_GLOBAL));
	cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&mix_wmma_C), sizeof(float) * M_GLOBAL * N_GLOBAL));

    int RAND = 0;

    if (RAND == 0)
    {
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_device_A), sizeof(float) * M_GLOBAL * K_GLOBAL));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_device_B), sizeof(float) * N_GLOBAL * K_GLOBAL));
        cudaErrCheck(hipMemcpy(ori_device_A, ori_host_A, sizeof(float) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
        cudaErrCheck(hipMemcpy(ori_device_B, ori_host_B, sizeof(float) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
        convertFp32ToFp16 <<< (M_GLOBAL * K_GLOBAL + 255) / 256, 256 >>> (ori_wmma_A, ori_device_A, M_GLOBAL * K_GLOBAL);
        convertFp32ToFp16 <<< (N_GLOBAL * K_GLOBAL + 255) / 256, 256 >>> (ori_wmma_B, ori_device_B, N_GLOBAL * K_GLOBAL);
    }
    else if (RAND = 1)
    {
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_host_A), sizeof(float) * M_GLOBAL * K_GLOBAL));
        cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&ori_host_B), sizeof(float) * N_GLOBAL * K_GLOBAL));
        curandErrCheck(hiprandGenerateUniform(gen, ori_host_A, M_GLOBAL * K_GLOBAL));
        curandErrCheck(hiprandGenerateUniform(gen, ori_host_B, N_GLOBAL * K_GLOBAL));
        convertFp32ToFp16 <<< (M_GLOBAL * K_GLOBAL + 255) / 256, 256 >>> (ori_wmma_A, ori_host_A, M_GLOBAL * K_GLOBAL);
        convertFp32ToFp16 <<< (N_GLOBAL * K_GLOBAL + 255) / 256, 256 >>> (ori_wmma_B, ori_host_B, N_GLOBAL * K_GLOBAL);
    }

    
	cudaErrCheck(hipMemcpy(mix_wmma_A, ori_wmma_A, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyDeviceToDevice));
	cudaErrCheck(hipMemcpy(mix_wmma_B, ori_wmma_B, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyDeviceToDevice));

	assert(((unsigned long long)ori_wmma_A) % 128 == 0);
	assert(((unsigned long long)ori_wmma_B) % 128 == 0);
	assert(((unsigned long long)ori_wmma_C) % 128 == 0);
    assert(((unsigned long long)mix_wmma_A) % 128 == 0);
	assert(((unsigned long long)mix_wmma_B) % 128 == 0);
	assert(((unsigned long long)mix_wmma_C) % 128 == 0);

	// cudaErrCheck(hipMemcpy(ori_wmma_A, ori_host_A, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
	// cudaErrCheck(hipMemcpy(mix_wmma_A, ori_host_A, sizeof(half) * M_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
	// cudaErrCheck(hipMemcpy(ori_wmma_B, ori_host_B, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
	// cudaErrCheck(hipMemcpy(mix_wmma_B, ori_host_B, sizeof(half) * N_GLOBAL * K_GLOBAL, hipMemcpyHostToDevice));
	cudaErrCheck(hipMemset(ori_wmma_C, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));
	cudaErrCheck(hipMemset(mix_wmma_C, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

    // start solo running 
    // ----------------------------------------------------------------------------------------------------------------------

    dim3 wmma_grid;
    dim3 wmma_block;
	wmma_grid.x = 68 * 4;
	wmma_block.x = THREADS_PER_BLOCK;
	int wmma_grid_dim_x = (M_TILES * N_TILES) / (BLOCK_COL_TILES * BLOCK_ROW_TILES);
	int wmma_block_dim_x = wmma_block.x;

    printf("[ORI] Running with tzgemm...\n");
    printf("[ORI] wmma_grid -- %d * 1 wmma_block -- %d * 1 \n", wmma_grid.x, wmma_block.x);

    checkKernelErrors((pers_tzgemm<<<wmma_grid, wmma_block>>>(ori_wmma_A, ori_wmma_B, ori_wmma_C,
                            64, 64, 64,
							4, wmma_block_dim_x, 1)));
	cudaErrCheck(hipMemset(ori_wmma_C, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));

    cudaErrCheck(hipEventRecord(startKERNEL));
    for(int i = 0; i < wmma_iter; i++) {
        checkKernelErrors((pers_tzgemm<<<wmma_grid, wmma_block>>>(ori_wmma_A, ori_wmma_B, ori_wmma_C,
                                M_GLOBAL, N_GLOBAL, K_GLOBAL,
                                wmma_grid_dim_x, wmma_block_dim_x, 1)));
    }
    cudaErrCheck(hipEventRecord(stopKERNEL));
    cudaErrCheck(hipEventSynchronize(stopKERNEL));
    cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
    printf("[ORI] tzgemm took %f us\n", kernel_time * 1000 / wmma_iter);

    hipblasHandle_t cublasHandle;
	cublasErrCheck(hipblasCreate(&cublasHandle));
	cublasErrCheck(hipblasSetMathMode(cublasHandle, HIPBLAS_TENSOR_OP_MATH));
	printf("Running with cuBLAS...\n");

    cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
					64, 64, 64, 
					&alpha_g,
					mix_wmma_B, HIP_R_16F, 64,
					mix_wmma_A, HIP_R_16F, 64,
					&beta_g, 
					mix_wmma_C, HIP_R_32F, 64,
					HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
	cudaErrCheck(hipMemset(mix_wmma_C, 0, sizeof(float) * M_GLOBAL * N_GLOBAL));


    cudaErrCheck(hipEventRecord(startKERNEL));
    for(int i = 0; i < wmma_iter; i++) {
        cublasErrCheck(hipblasGemmEx(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_N, 
                        N_GLOBAL, M_GLOBAL, K_GLOBAL, 
                        &alpha_g,
                        mix_wmma_B, HIP_R_16F, K_GLOBAL,
                        mix_wmma_A, HIP_R_16F, K_GLOBAL,
                        &beta_g, 
                        mix_wmma_C, HIP_R_32F, N_GLOBAL,
                        HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP));
    }
    cudaErrCheck(hipEventRecord(stopKERNEL));
    cudaErrCheck(hipEventSynchronize(stopKERNEL));
    cudaErrCheck(hipEventElapsedTime(&kernel_time, startKERNEL, stopKERNEL));
    printf("[ORI] cublas took %f us\n", kernel_time * 1000 /wmma_iter);
    

    printf("Checking results...\n");
    cudaErrCheck(hipMemcpy(ori_result_C, ori_wmma_C, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));
    cudaErrCheck(hipMemcpy(mix_result_C, mix_wmma_C, sizeof(float) * M_GLOBAL * N_GLOBAL, hipMemcpyDeviceToHost));

    int errors = 0;
    for (int i = 0; i < M_GLOBAL * N_GLOBAL; i++) {
        float v1 = ori_result_C[i];
        float v2 = mix_result_C[i];
        if (fabs(v1 - v2) > 0.00001f) {
            errors++;
            if (errors < 5) printf("%f %f\n", v1, v2);
        }
		if (i < 10) printf("%f %f\n", ori_result_C[i], mix_result_C[i]);
    }
    if (errors > 0) {
        printf("[WMMA] ORIGIN VERSION does not agree with MY VERSION! %d errors!\n", errors);
    }
    else {
        printf("[WMMA] Results verified: ORIGIN VERSION and MY VERSION agree.\n");
    }
    cudaErrCheck(hipEventDestroy(startKERNEL));
    cudaErrCheck(hipEventDestroy(stopKERNEL));
    
    cudaErrCheck(hipDeviceReset());
    return 0;
}
